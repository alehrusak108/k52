#include "hip/hip_runtime.h"
#include <k52/dsp/transform/cuda_fast_fourier_transform.h>
#include <k52/dsp/transform/util/cuda_utils.h>
#include <cstdio>
#include <fstream>
#include <stdexcept>

#ifdef BUILD_WITH_CUDA

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <boost/thread/mutex.hpp>
#include <boost/smart_ptr/make_shared.hpp>
#include "../../../../../../../usr/local/cuda/include/hipfft/hipfft.h"
#include "../../../../../../../usr/local/cuda/include/hipfft/hipfftXt.h"
#include "../../../../../../../usr/local/cuda/include/hip/hip_runtime_api.h"

// TODO: DELETE THIS IMPORTS!

#endif

using ::std::vector;
using ::std::complex;
using ::std::invalid_argument;
using ::std::runtime_error;

namespace k52
{
namespace dsp
{

#ifdef BUILD_WITH_CUDA

// Using pImpl approach to hide CUFFT from outside use
// NOTE: Prefix "device_" means that variable is allocated in CUDA Device Memory
//       Prefix "host_" means that variable is allocated in RAM (Host)

class CudaFastFourierTransform::CudaFastFourierTransformImpl
{

public:
    CudaFastFourierTransformImpl(size_t sequence_size, int transforms_count)
            : signal_size_(sequence_size), transforms_count_(transforms_count) {

        std::ofstream test_output;
        test_output.open("test_output.txt");

        boost::mutex::scoped_lock scoped_lock(cuda_mutex_);

        if (sequence_size <= 0) {
            throw std::invalid_argument("sequence_size <= 0");
        }

        signal_memory_size_ = sizeof(hipfftComplex) * signal_size_;

        // Use only 2 GPUs if even more available
        int available_gpus;
        hipGetDeviceCount(&available_gpus);
        int gpu_to_use = available_gpus > 2 ? 2 : available_gpus;
        int *gpu_array = GetAvailableGPUArray(gpu_to_use);

        cufft_work_size_ = (size_t *) malloc (sizeof(size_t) * gpu_to_use);
        hipfftCreate(&cufft_execution_plan_);
        hipfftResult set_gpus_result = hipfftXtSetGPUs(cufft_execution_plan_, available_gpus, gpu_array);
        test_output << std::endl << "CUFFT Set GPUs result: " << set_gpus_result << std::endl;
        hipfftResult plan_prepare_result = hipfftMakePlan1d(
                cufft_execution_plan_,
                signal_size_,
                HIPFFT_C2C,
                transforms_count_,
                cufft_work_size_
        );
        test_output << std::endl << "CUFFT Execution Plan prepared: " << plan_prepare_result << std::endl;
    }

    ~CudaFastFourierTransformImpl() {

        std::ofstream test_output;
        test_output.open("test_output.txt");
        test_output << "Destroying CUFFT Context..." << std::endl;

        // Destroy CUFFT Execution Plan
        hipfftResult destructor_result = hipfftDestroy(cufft_execution_plan_);
        std::cout << "CUFFT Execution Plan destructor returned: " << destructor_result << std::endl << std::endl;

        free(cufft_work_size_);

        boost::mutex::scoped_lock scoped_lock(cuda_mutex_);
        test_output.close();
    }

    vector<complex<double> > DirectTransform(const vector<complex<double> > &sequence)
    {
        return Transform(sequence, HIPFFT_FORWARD);
    }

    vector<complex<double> > InverseTransform(const vector<complex<double> > &sequence)
    {
        return Transform(sequence, HIPFFT_BACKWARD);
    }

    vector<complex<double> > Transform(const vector<complex<double> > &sequence, int transform_direction) const
    {

        std::ofstream test_output;
        test_output.open("test_output.txt");
        
        if (signal_size_ != sequence.size()) {
            throw std::invalid_argument(
                    "CudaFastFourierTransform can transform only data of the same size as was specified on construction.");
        }

        hipfftComplex *host_signal = CudaUtils::VectorToCufftComplex(sequence);

        hipLibXtDesc *device_signal;
        hipfftXtMalloc(cufft_execution_plan_, &device_signal, HIPFFT_XT_FORMAT_INPLACE);
        hipfftXtMemcpy(cufft_execution_plan_, device_signal, host_signal, HIPFFT_COPY_HOST_TO_DEVICE);

        test_output << std::endl << "Signal memory allocated: " << signal_memory_size_ << " bytes." << std::endl;

        // NOTE: Transformed signal will be written instead of source signal to escape memory wasting
        clock_t execution_time = clock();
        hipfftResult execution_result = hipfftXtExecDescriptorC2C(
                cufft_execution_plan_,
                device_signal,
                device_signal,
                HIPFFT_FORWARD
        );
        test_output << std::endl << "CUFFT Transformation finished in: " << (float) (clock() - execution_time) / CLOCKS_PER_SEC << " seconds " << std::endl;
        test_output << std::endl << "CUFFT C2C (float) Execution result: " << execution_result << std::endl;

        // Copy Device memory (FFT calculation results - d_signal_output_) to Host memory (RAM)
        hipfftXtMemcpy(cufft_execution_plan_, host_signal, device_signal, HIPFFT_COPY_DEVICE_TO_HOST);

        vector<complex<double> > result_vector = CudaUtils::CufftComplexToVector(host_signal, signal_size_);

        hipfftXtFree(device_signal);
        hipFree(host_signal);

        test_output.close();
        return result_vector;
    }

private:

    // static fields and initializers
    static boost::mutex cuda_mutex_;

    // instance fields and initializers
    size_t signal_size_;
    int transforms_count_;
    int signal_memory_size_;

    size_t *cufft_work_size_;
    hipfftHandle cufft_execution_plan_;

    int* GetAvailableGPUArray(int gpu_count)
    {
        int *gpu_array = (int*) malloc(sizeof(int) * gpu_count);
        for (unsigned int index = 0; index < gpu_count; index++)
        {
            gpu_array[index] = index;
        }
        return gpu_array;
    }
};

boost::mutex CudaFastFourierTransform::CudaFastFourierTransformImpl::cuda_mutex_;

CudaFastFourierTransform::CudaFastFourierTransform(size_t sequence_size, int planned_executions)
{
    cuda_fast_fourier_transform_impl_ =
            boost::make_shared<CudaFastFourierTransformImpl>(sequence_size, planned_executions);
}

CudaFastFourierTransform::~CudaFastFourierTransform()
{
}

vector<complex<double> > CudaFastFourierTransform::DirectTransform(
        const vector<complex<double> > &sequence) const
{
    return cuda_fast_fourier_transform_impl_->DirectTransform(sequence);
}

vector<complex<double> > CudaFastFourierTransform::InverseTransform(
        const vector<complex<double> > &sequence) const
{
    return cuda_fast_fourier_transform_impl_->InverseTransform(sequence);
}

#endif //BUILD_WITH_CUDA

} // namespace dsp
} // namespace k52