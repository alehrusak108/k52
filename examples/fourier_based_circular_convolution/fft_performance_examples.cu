#include "hip/hip_runtime.h"
#include <k52/dsp/transform/i_fourier_transform.h>
#include <k52/dsp/transform/fast_fourier_transform.h>
#include <k52/dsp/transform/fourier_based_circular_convolution.h>
#include <k52/dsp/transform/circular_convolution.h>

#include <k52/common/helpers.h>

#include <boost/smart_ptr/shared_ptr.hpp>

#include <iostream>
#include <ctime>
#include <cstdlib>
#include <vector>
#include <fstream>

#ifdef BUILD_WITH_CUDA

#include <k52/dsp/transform/cuda_fast_fourier_transform.h>
#include <k52/dsp/transform/util/cuda_utils.h>

#endif

#define CUFFT_EXECUTIONS_PLANNED 1

#define PAGE_SIZE 8

using namespace std;

using namespace k52::dsp;
using namespace k52::common;

void CUFFTPerformanceTest(vector<complex<double> > input_signal)
{

    ofstream test_output;
    test_output.open("fast_fourier_transform_test.txt", ios::out | ios::app);
    test_output << endl << "[ CUFFT Performance TEST ] STARTED." << endl;

    clock_t planning_time = clock();
    CudaFastFourierTransform cufftTransformer(input_signal.size(), PAGE_SIZE);
    cufftTransformer.SetDeviceSignal(CudaUtils::VectorToCufftComplexAlloc(input_signal));
    test_output << "CUFFT Data Transfer and Execution Plan prepared in: " << (float) (clock() - planning_time) / CLOCKS_PER_SEC << " seconds" << endl;

    clock_t execution_time = clock();

    cufftTransformer.DirectTransform();
    vector<complex<double> > output = cufftTransformer.GetTransformResult();

    //Helpers::PrintComplexVector(output);

    clock_t finish = clock() - execution_time;
    test_output << endl << "Time elapsed for CUFFT Transform Test: " << (double) (clock() - execution_time) / CLOCKS_PER_SEC << " seconds " << endl << endl;
    test_output << "[ CUFFT Performance TEST ] FINISHED." << endl << endl;
    test_output.close();
}

void FFTWPerformanceTest(vector<complex<double> > input_signal)
{

    ofstream test_output;
    test_output.open("fast_fourier_transform_test.txt", ios::out | ios::app);
    test_output << "[ FFTW3 Performance TEST ] STARTED." << endl;

    int total_pages = input_signal.size() / PAGE_SIZE;

    clock_t planning_time = clock();
    FastFourierTransform fftwTransformer(PAGE_SIZE);
    test_output << endl << "FFTW3 Execution Plan prepared in: " << (float) (clock() - planning_time) / CLOCKS_PER_SEC << " seconds" << endl;

    clock_t execution_time = clock();
    for (size_t page_number = 0; page_number < total_pages; page_number++)
    {
        size_t start_index = PAGE_SIZE * page_number;
        size_t end_index = start_index + PAGE_SIZE;
        vector<complex<double> >::const_iterator page_start = input_signal.begin() + start_index;
        vector<complex<double> >::const_iterator page_end = input_signal.begin() + end_index;
        vector<complex<double> > signal_page(page_start, page_end);

        vector<complex<double> > output = fftwTransformer.DirectTransform(signal_page);
    }

    test_output << endl << endl << "Time elapsed for FFTW3 Transform Test: " << (double) (clock() - execution_time) / CLOCKS_PER_SEC << " seconds " << endl << endl;

    test_output << "[ FFTW3 Performance TEST ] FINISHED." << endl << endl;
    test_output.close();
}

int main(int argc, char* argv[])
{
    srand(time(NULL));

    ofstream test_output;
    test_output.open("fast_fourier_transform_test.txt", ios::out | ios::app);
    test_output << endl << "FFT PERFORMANCE TEST (FFTW vs CUDA)" << endl << endl;
    int signal_size = 64;
    //for (int test_index = 1; test_index <= 7; test_index++) {
        vector<complex<double> > input_signal = Helpers::GenerateComplexSignal(signal_size);
        //test_output << endl << "TEST #" << test_index << "\t" << "Signal Length is: " << signal_size << endl;
        CUFFTPerformanceTest(input_signal);
        test_output << "-----------------------------------------------------------------------" << endl << endl;
        //FFTWPerformanceTest(input_signal);
        test_output << "===============================================================================" << endl << endl;
        signal_size *= 2;
    //}
    test_output.close();
}