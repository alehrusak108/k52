#include <k52/dsp/transform/cuda_fast_fourier_transform.h>
#include <k52/dsp/transform/util/cuda_utils.h>
#include <cstdio>
#include <fstream>
#include <stdexcept>

#ifdef BUILD_WITH_CUDA

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <boost/thread/mutex.hpp>
#include <boost/smart_ptr/make_shared.hpp>
#include "../../../../../../../usr/local/cuda/include/hipfft/hipfft.h"
#include "../../../../../../../usr/local/cuda/include/hipfft/hipfftXt.h"
#include "../../../../../../../usr/local/cuda/include/hip/hip_runtime_api.h"

// TODO: DELETE THIS IMPORTS!

#endif

using ::std::vector;
using ::std::complex;
using ::std::invalid_argument;
using ::std::runtime_error;

namespace k52
{
namespace dsp
{

#ifdef BUILD_WITH_CUDA

// Using pImpl approach to hide CUFFT from outside use
// NOTE: Prefix "device_" means that variable is allocated in CUDA Device Memory
//       Prefix "host_" means that variable is allocated in RAM (Host)

class CudaFastFourierTransform::CudaFastFourierTransformImpl
{

public:
    CudaFastFourierTransformImpl(vector<complex<double> > signal, size_t page_size)
            : signal_(signal), page_size_(page_size) {

        boost::mutex::scoped_lock scoped_lock(cuda_mutex_);

        if (signal_.size() % page_size != 0)
        {
            throw std::invalid_argument("CUDA FFT FATAL: Modulo of sequence_size with page_size should be 0.");
        }

        signal_size_ = signal_.size();
        signal_memory_size_ = sizeof(hipfftComplex) * signal_size_;
        total_pages_ = signal_size_ / page_size_;

        std::cout << std::endl << "Constructing the CUFFT Context with the following parameters: " << std::endl
                  << "Signal Size: " << signal_size_ << std::endl
                  << "Page Size: " << page_size_ << std::endl
                  << "Total Pages: " << total_pages_ << std::endl << std::endl;

        hipfftResult hipfftResult;
        hipfftResult = hipfftPlan1d(&cufft_execution_plan_, page_size_, HIPFFT_C2C, BATCH_COUNT_);
        CudaUtils::checkCufftErrors(hipfftResult, "CUFFT Create Plan");

        for (unsigned int page_number = 0; page_number < total_pages_; page_number++)
        {
            size_t start_index = page_size_ * page_number;
            size_t end_index = start_index + page_size_ - 1;
            vector<complex<double> >::const_iterator page_start = signal_.begin() + start_index;
            vector<complex<double> >::const_iterator page_end = signal_.begin() + end_index;
            vector<complex<double> > signal_page(page_start, page_end);

            device_signal_pages_ = (hipfftComplex **) malloc(sizeof(hipfftComplex) * page_size_);

            hipError_t cuda_result;
            cuda_result = hipMalloc((void **) &device_signal_pages_[page_number], sizeof(hipfftComplex) * page_size_);
            CudaUtils::checkErrors(cuda_result, "CUFFT FORWARD allocation on single GPU");

            // Copy the whole signal to Device
            host_signal_page_ = CudaUtils::VectorToCufftComplex(signal_page);
            cuda_result = hipMemcpy(device_signal_pages_[page_number], host_signal_page_, page_size_, hipMemcpyHostToDevice);
            CudaUtils::checkErrors(cuda_result, "CUFFT FORWARD memory copying from Host to Device");
        }
    }

    ~CudaFastFourierTransformImpl() {

        std::cout << "Destroying CUFFT Context..." << std::endl << std::endl;

        hipfftResult cufft_result = hipfftDestroy(cufft_execution_plan_);
        CudaUtils::checkCufftErrors(cufft_result, "CUFFT Execution Plan destructor");

        hipError_t cuda_result = hipFree(device_signal_pages_);
        CudaUtils::checkErrors(cuda_result, "CUFFT Execution Plan destructor");

        free(host_signal_page_);

        boost::mutex::scoped_lock scoped_lock(cuda_mutex_);

        std::cout << "CUFFT Context Destroyed" << std::endl << std::endl;
    }

    void DirectTransform()
    {
        Transform(HIPFFT_FORWARD);
    }

    void InverseTransform()
    {
        Transform(HIPFFT_BACKWARD);
    }

    void Transform(int transform_direction)
    {
        hipfftResult cufft_result;

        for (unsigned int page_number = 0; page_number < total_pages_; page_number++)
        {
            // NOTE: Transformed signal will be written instead of source signal to escape memory wasting
            cufft_result = hipfftExecC2C(
                    cufft_execution_plan_,
                    device_signal_pages_[page_number],
                    device_signal_pages_[page_number],
                    transform_direction
            );
            CudaUtils::checkCufftErrors(cufft_result, "CUFFT FORWARD C2C execution");
        }
    }

    vector<complex<double> > GetTransformResult()
    {
        vector<complex<double> > result(signal_size_);
        for (unsigned int page_number = 0; page_number < total_pages_; page_number++)
        {
            hipError_t cuda_result;
            cuda_result = hipMemcpy(host_signal_page_, device_signal_pages_[page_number], page_size_, hipMemcpyDeviceToHost);
            CudaUtils::checkErrors(cuda_result, "CUFFT FORWARD C2C Copying execution results from Device to Host");
        }
        return result;
    }

private:

    // static fields and initializers
    static boost::mutex cuda_mutex_;
    static const int BATCH_COUNT_ = 1;

    // instance fields and initializers
    vector<complex<double> > signal_;
    size_t signal_size_;
    size_t page_size_;
    int total_pages_;
    int signal_memory_size_;

    hipfftComplex **device_signal_pages_;
    hipfftComplex *host_signal_page_;
    hipfftHandle cufft_execution_plan_;
};

boost::mutex CudaFastFourierTransform::CudaFastFourierTransformImpl::cuda_mutex_;

CudaFastFourierTransform::CudaFastFourierTransform(vector<complex<double> > signal, size_t page_size)
{
    cuda_fast_fourier_transform_impl_ =
            boost::make_shared<CudaFastFourierTransformImpl>(signal, page_size);
}

CudaFastFourierTransform::~CudaFastFourierTransform()
{
}

void CudaFastFourierTransform::DirectTransform()
{
    cuda_fast_fourier_transform_impl_->DirectTransform();
}

void CudaFastFourierTransform::InverseTransform()
{
    cuda_fast_fourier_transform_impl_->InverseTransform();
}

vector<complex<double> > CudaFastFourierTransform::GetTransformResult()
{
    return cuda_fast_fourier_transform_impl_->GetTransformResult();
}

#endif //BUILD_WITH_CUDA

} // namespace dsp
} // namespace k52