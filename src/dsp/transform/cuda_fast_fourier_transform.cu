#include "hip/hip_runtime.h"
#include <k52/dsp/transform/cuda_fast_fourier_transform.h>
#include <k52/dsp/transform/util/cuda_utils.h>
#include <cstdio>
#include <fstream>
#include <stdexcept>

#ifdef BUILD_WITH_CUDA

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <boost/thread/mutex.hpp>
#include <boost/smart_ptr/make_shared.hpp>
#include <k52/common/helpers.h>
#include "../../../../../../../usr/local/cuda/include/hipfft/hipfft.h"
#include "../../../../../../../usr/local/cuda/include/hipfft/hipfftXt.h"
#include "../../../../../../../usr/local/cuda/include/hip/hip_runtime_api.h"
#include "../../../../../../../usr/local/cuda/include/"

// TODO: DELETE THIS IMPORTS!

#endif

using ::std::ios;
using ::std::ofstream;
using ::std::vector;
using ::std::complex;
using ::std::invalid_argument;
using ::std::runtime_error;

namespace k52
{
namespace dsp
{

#ifdef BUILD_WITH_CUDA

// Initializes given pointer to signal page with signal data using "begin" and "end" indexes
__global__ void InitializeSignalPage(hipfftComplex *page, hipfftComplex *signal, int page_size, int from_index)
{
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < page_size)
    {
        page[thread_id].x = signal[from_index + thread_id].x;
        page[thread_id].y = signal[from_index + thread_id].y;
    }
}

// Copies given pointer to signal page into signal using "begin" and "end" indexes
__global__ void CopyPageToSignal(hipfftComplex *signal, hipfftComplex *page, int page_size, int from_index)
{
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < page_size)
    {
        signal[from_index + thread_id].x = page[thread_id].x;
        signal[from_index + thread_id].y = page[thread_id].y;
    }
}

// Using pImpl approach to hide CUFFT from outside use
// NOTE: Prefix "device_" means that variable is allocated in CUDA Device Memory
//       Prefix "host_" means that variable is allocated in RAM (Host)
class CudaFastFourierTransform::CudaFastFourierTransformImpl
{

public:
    CudaFastFourierTransformImpl(size_t signal_size, size_t page_size)
            : signal_size_(signal_size), page_size_(page_size)
    {

        ofstream test_output;
        test_output.open("fast_fourier_transform_test.txt", ios::out | ios::app);

        boost::mutex::scoped_lock scoped_lock(cuda_mutex_);

        if (signal_size <= 0)
        {
            throw std::invalid_argument("CUDA FFT FATAL: Modulo of sequence_size with page_size should be 0.");
        }

        total_pages_ = signal_size_ / page_size_;

        hipSetDevice(0);

        test_output << std::endl << "Constructing the CUFFT Context with the following parameters: " << std::endl
                  << "Signal Size: " << signal_size_ << std::endl
                  << "Page Size: " << page_size_ << std::endl
                  << "Total Pages: " << total_pages_ << std::endl << std::endl;

        hipfftResult hipfftResult;
        hipfftResult = hipfftPlan1d(&cufft_execution_plan_, page_size_, HIPFFT_C2C, BATCH_COUNT_);
        CudaUtils::checkCufftErrors(hipfftResult, "CUFFT Create Plan");

        hipError_t cuda_result;
        cuda_result = hipMalloc((void **) &device_signal_, sizeof(hipfftComplex) * signal_size_);
        CudaUtils::checkErrors(cuda_result, "CUFFT FORWARD allocation on single GPU");

        cuda_result = hipMalloc((void **) &device_signal_page_, sizeof(hipfftComplex) * page_size_);
        CudaUtils::checkErrors(cuda_result, "CUFFT FORWARD allocation memory for a signal page");

        host_signal_ = (hipfftComplex *) malloc(sizeof(hipfftComplex) * signal_size_);
    }

    ~CudaFastFourierTransformImpl() {

        ofstream test_output;
        test_output.open("fast_fourier_transform_test.txt", ios::out | ios::app);

        test_output << "Destroying CUFFT Context..." << std::endl << std::endl;

        hipfftResult cufft_result = hipfftDestroy(cufft_execution_plan_);
        CudaUtils::checkCufftErrors(cufft_result, "CUFFT Execution Plan destructor");

        hipError_t cuda_result;
        cuda_result = hipFree(device_signal_);
        CudaUtils::checkErrors(cuda_result, "CUFFT hipFree for device_signal_");

        cuda_result = hipFree(device_signal_page_);
        CudaUtils::checkErrors(cuda_result, "CUFFT hipFree for device_signal_page_");

        free(host_signal_);

        boost::mutex::scoped_lock scoped_lock(cuda_mutex_);

        test_output << "CUFFT Context Destroyed" << std::endl << std::endl;
    }

    void DirectTransform()
    {
        Transform(HIPFFT_FORWARD);
    }

    void InverseTransform()
    {
        Transform(HIPFFT_BACKWARD);
    }

    void Transform(int transform_direction)
    {
        // MAKE device_signal 1D and copy arrays in __global__ function
        for (size_t page_number = 0; page_number < total_pages_; page_number++)
        {
            size_t from_index = page_size_ * page_number;
            InitializeSignalPage<<<256, 512>>>(device_signal_page_, device_signal_, page_size_, from_index);

            hipfftResult cufft_result = hipfftExecC2C(
                    cufft_execution_plan_,
                    device_signal_page_,
                    device_signal_page_,
                    transform_direction
            );
            CudaUtils::checkCufftErrors(cufft_result, "CUFFT FORWARD C2C execution");

            CopyPageToSignal<<<256, 512>>>(device_signal_, device_signal_page_, page_size_, from_index);
        }
    }

    void SetDeviceSignal(vector<complex<double> > signal)
    {
        // Copy the whole signal to Device
        CudaUtils::VectorToCufftComplex(signal, host_signal_);
        hipError_t cuda_result = hipMemcpy(device_signal_, host_signal_, sizeof(hipfftComplex) * signal_size_, hipMemcpyHostToDevice);
        CudaUtils::checkErrors(cuda_result, "CUFFT SetDeviceSignal setting other signal. Copy from Host to Device");
    }

    void SetDeviceSignal(hipfftComplex *signal)
    {
        // Copy the whole signal to Device
        hipError_t cuda_result = hipMemcpy(device_signal_, signal, sizeof(hipfftComplex) * signal_size_, hipMemcpyHostToDevice);
        CudaUtils::checkErrors(cuda_result, "CUFFT SetDeviceSignal setting other signal. Copy from Host to Device");
    }

    vector<complex<double> > GetTransformResult()
    {
        hipError_t cuda_result = hipMemcpy(host_signal_, device_signal_, signal_size_ * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
        for (int i = 0; i < signal_size_; i++)
        {
            std::cout << host_signal_[i].x << "\t" << host_signal_[i].y << std::endl;
        }
        CudaUtils::checkErrors(cuda_result, "CUFFT GetTransformResult Copying execution results from Device to Host");
        return CudaUtils::CufftComplexToVector(host_signal_, signal_size_);
    }

    hipfftComplex* GetTransformResultArray()
    {
        hipError_t cuda_result = hipMemcpy(host_signal_, device_signal_, signal_size_ * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
        CudaUtils::checkErrors(cuda_result, "CUFFT GetTransformResultArray Copying execution results from Device to Host");
        return host_signal_;
    }

private:

    // static fields and initializers
    static boost::mutex cuda_mutex_;
    static const int BATCH_COUNT_ = 1;

    // instance fields and initializers
    size_t signal_size_;
    size_t page_size_;
    int total_pages_;

    hipfftComplex *device_signal_;
    hipfftComplex *host_signal_;
    hipfftComplex *device_signal_page_;
    hipfftHandle cufft_execution_plan_;
};

boost::mutex CudaFastFourierTransform::CudaFastFourierTransformImpl::cuda_mutex_;

CudaFastFourierTransform::CudaFastFourierTransform(size_t signal_size, size_t page_size)
{
    cuda_fast_fourier_transform_impl_ =
            boost::make_shared<CudaFastFourierTransformImpl>(signal_size, page_size);
}

CudaFastFourierTransform::~CudaFastFourierTransform()
{
}

void CudaFastFourierTransform::DirectTransform()
{
    cuda_fast_fourier_transform_impl_->DirectTransform();
}

void CudaFastFourierTransform::InverseTransform()
{
    cuda_fast_fourier_transform_impl_->InverseTransform();
}

void CudaFastFourierTransform::SetDeviceSignal(hipfftComplex *signal)
{
    cuda_fast_fourier_transform_impl_->SetDeviceSignal(signal);
}

void CudaFastFourierTransform::SetDeviceSignal(vector<complex<double> > signal)
{
    cuda_fast_fourier_transform_impl_->SetDeviceSignal(signal);
}

vector<complex<double> > CudaFastFourierTransform::GetTransformResult()
{
    return cuda_fast_fourier_transform_impl_->GetTransformResult();
}

hipfftComplex *CudaFastFourierTransform::GetTransformResultArray()
{
    return cuda_fast_fourier_transform_impl_->GetTransformResultArray();
}

#endif //BUILD_WITH_CUDA

} // namespace dsp
} // namespace k52