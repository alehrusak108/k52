#include "hip/hip_runtime.h"
#include <k52/dsp/transform/cuda_fast_fourier_transform.h>
#include <k52/dsp/transform/util/cuda_utils.h>
#include <cstdio>
#include <fstream>
#include <stdexcept>

#ifdef BUILD_WITH_CUDA

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <boost/thread/mutex.hpp>
#include <boost/smart_ptr/make_shared.hpp>
#include "../../../../../../../usr/local/cuda/include/hipfft/hipfft.h"
#include "../../../../../../../usr/local/cuda/include/hipfft/hipfftXt.h"
#include "../../../../../../../usr/local/cuda/include/hip/hip_runtime_api.h"

// TODO: DELETE THIS IMPORTS!

#endif

using ::std::vector;
using ::std::complex;
using ::std::invalid_argument;
using ::std::runtime_error;

namespace k52
{
namespace dsp
{

#ifdef BUILD_WITH_CUDA

// Using pImpl approach to hide CUFFT from outside use
// NOTE: Prefix "device_" means that variable is allocated in CUDA Device Memory
//       Prefix "host_" means that variable is allocated in RAM (Host)

class CudaFastFourierTransform::CudaFastFourierTransformImpl
{

public:
    CudaFastFourierTransformImpl(vector<complex<double> > signal, size_t page_size)
            : signal_(signal), page_size_(page_size) {

        boost::mutex::scoped_lock scoped_lock(cuda_mutex_);

        if (signal_.size() % page_size != 0)
        {
            throw std::invalid_argument("CUDA FFT FATAL: Modulo of sequence_size with page_size should be 0.");
        }

        signal_size_ = signal_.size();
        signal_memory_size_ = sizeof(hipfftComplex) * signal_size_;
        total_pages = signal_size_ / page_size_;

        hipfftResult hipfftResult;
        hipfftResult = hipfftPlan1d(&cufft_execution_plan_, page_size_, HIPFFT_C2C, BATCH_COUNT_);
        CudaUtils::checkCufftErrors(hipfftResult, "CUFFT Create Plan");

        hipError_t cuda_result;
        cuda_result = hipMalloc((void **) &device_signal_, signal_memory_size_);
        CudaUtils::checkErrors(cuda_result, "CUFFT FORWARD allocation on single GPU");

        // Copy the whole signal to Device
        host_signal_ = CudaUtils::VectorToCufftComplex(signal_);
        cuda_result = hipMemcpy(device_signal_, host_signal_, signal_size_, hipMemcpyHostToDevice);
        CudaUtils::checkErrors(cuda_result, "CUFFT FORWARD memory copying from Host to Device");
    }

    ~CudaFastFourierTransformImpl() {

        std::cout << "Destroying CUFFT Context..." << std::endl << std::endl;

        hipfftResult cufft_result = hipfftDestroy(cufft_execution_plan_);
        CudaUtils::checkCufftErrors(cufft_result, "CUFFT Execution Plan destructor");

        hipError_t cuda_result = hipFree(device_signal_);
        CudaUtils::checkErrors(cuda_result, "CUFFT Execution Plan destructor");

        free(host_signal_);

        boost::mutex::scoped_lock scoped_lock(cuda_mutex_);

        std::cout << "CUFFT Context Destroyed" << std::endl << std::endl;
    }

    void DirectTransform()
    {
        Transform(HIPFFT_FORWARD);
    }

    void InverseTransform()
    {
        Transform(HIPFFT_BACKWARD);
    }

    void Transform(int transform_direction)
    {
        hipfftResult cufft_result;

        for (unsigned int page_number = 0; page_number < total_pages; page_number++) {
            size_t start_index = page_size_ * page_number;
            size_t end_index = start_index + page_size_;
            vector<complex<double> >::const_iterator page_start = signal_.begin() + start_index;
            vector<complex<double> >::const_iterator page_end = signal_.begin() + end_index;
            vector<complex<double> > signal_page(page_start, page_end);

            // NOTE: Transformed signal will be written instead of source signal to escape memory wasting
            clock_t execution_time = clock();
            cufft_result = hipfftExecC2C(
                    cufft_execution_plan_,
                    (device_signal_ + start_index),
                    (device_signal_ + end_index),
                    transform_direction
            );
            std::cout << std::endl << "CUFFT FORWARD Transformation finished in: " << (float) (clock() - execution_time) / CLOCKS_PER_SEC << " seconds " << std::endl;
            CudaUtils::checkCufftErrors(cufft_result, "CUFFT FORWARD C2C execution");
        }
    }

    vector<complex<double> > GetTransformResult() const
    {
        // Copy whole device memory (FFT calculation results - device_signal) to Host memory (RAM)
        hipError_t cuda_result;
        cuda_result = hipMemcpy(host_signal_, device_signal_, page_size_, hipMemcpyDeviceToHost);
        CudaUtils::checkErrors(cuda_result, "CUFFT FORWARD C2C Copying execution results from Device to Host");

        return CudaUtils::CufftComplexToVector(host_signal_, signal_size_);
    }

private:

    // static fields and initializers
    static boost::mutex cuda_mutex_;
    static const int BATCH_COUNT_ = 1;

    // instance fields and initializers
    vector<complex<double> > signal_;
    size_t signal_size_;
    size_t page_size_;
    int total_pages;
    int signal_memory_size_;

    hipfftComplex *device_signal_;
    hipfftComplex *host_signal_;
    hipfftHandle cufft_execution_plan_;
};

boost::mutex CudaFastFourierTransform::CudaFastFourierTransformImpl::cuda_mutex_;

CudaFastFourierTransform::CudaFastFourierTransform(vector<complex<double> > signal, size_t page_size)
{
    cuda_fast_fourier_transform_impl_ =
            boost::make_shared<CudaFastFourierTransformImpl>(signal, page_size);
}

CudaFastFourierTransform::~CudaFastFourierTransform()
{
}

void CudaFastFourierTransform::DirectTransform()
{
    cuda_fast_fourier_transform_impl_->DirectTransform();
}

void CudaFastFourierTransform::InverseTransform()
{
    cuda_fast_fourier_transform_impl_->InverseTransform();
}

vector<complex<double> > CudaFastFourierTransform::GetTransformResult() const
{
    return cuda_fast_fourier_transform_impl_->GetTransformResult();
}

#endif //BUILD_WITH_CUDA

} // namespace dsp
} // namespace k52