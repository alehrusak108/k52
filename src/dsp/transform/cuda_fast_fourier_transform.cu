#include "hip/hip_runtime.h"
#include <k52/dsp/transform/cuda_fast_fourier_transform.h>
#include <k52/dsp/transform/util/cuda_utils.h>
#include <cstdio>
#include <fstream>
#include <stdexcept>

#ifdef BUILD_WITH_CUDA

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <boost/thread/mutex.hpp>
#include <boost/smart_ptr/make_shared.hpp>
#include "../../../../../../../usr/local/cuda/include/hipfft/hipfft.h"
#include "../../../../../../../usr/local/cuda/include/hipfft/hipfftXt.h"
#include "../../../../../../../usr/local/cuda/include/hip/hip_runtime_api.h"

// TODO: DELETE THIS IMPORTS!

#endif

using ::std::vector;
using ::std::complex;
using ::std::invalid_argument;
using ::std::runtime_error;

namespace k52
{
namespace dsp
{

#ifdef BUILD_WITH_CUDA

// Using pImpl approach to hide CUFFT from outside use
// NOTE: Prefix "device_" means that variable is allocated in CUDA Device Memory
//       Prefix "host_" means that variable is allocated in RAM (Host)

class CudaFastFourierTransform::CudaFastFourierTransformImpl
{

public:
    CudaFastFourierTransformImpl(size_t sequence_size, int transforms_count)
            : signal_size_(sequence_size), transforms_count_(transforms_count) {

        boost::mutex::scoped_lock scoped_lock(cuda_mutex_);

        if (sequence_size <= 0)
        {
            throw std::invalid_argument("sequence_size <= 0");
        }

        signal_memory_size_ = sizeof(hipfftComplex) * signal_size_;

        // Use only 2 GPUs if even more available
        hipError_t error = hipGetDeviceCount(&available_gpus);
        CudaUtils::checkErrors(error, "CUDA Get Device Count");

        int *gpu_array = GetAvailableGPUArray();

        cufft_work_size_ = (size_t *) malloc (sizeof(size_t) * available_gpus);
        hipfftResult result;
        result = hipfftCreate(&cufft_execution_plan_);
        CudaUtils::checkCufftErrors(result, "CUFFT Create Plan");

        result = hipfftXtSetGPUs(cufft_execution_plan_, available_gpus, gpu_array);
        CudaUtils::checkCufftErrors(result, "CUFFT Set GPUs");

        result = hipfftMakePlan1d(
                cufft_execution_plan_,
                signal_size_,
                HIPFFT_C2C,
                transforms_count_,
                cufft_work_size_
        );
        CudaUtils::checkCufftErrors(result, "CUFFT Execution Plan preparing");
    }

    ~CudaFastFourierTransformImpl() {

        std::cout << "Destroying CUFFT Context..." << std::endl << std::endl;

        hipfftResult result = hipfftDestroy(cufft_execution_plan_);
        CudaUtils::checkCufftErrors(result, "CUFFT Execution Plan destructor");

        free(cufft_work_size_);

        boost::mutex::scoped_lock scoped_lock(cuda_mutex_);

        std::cout << "CUFFT Context Destroyed" << std::endl << std::endl;
    }

    vector<complex<double> > DirectTransform(const vector<complex<double> > &sequence)
    {
        return Transform(sequence, HIPFFT_FORWARD);
    }

    vector<complex<double> > InverseTransform(const vector<complex<double> > &sequence)
    {
        return Transform(sequence, HIPFFT_BACKWARD);
    }

    vector<complex<double> > Transform(const vector<complex<double> > &sequence, int transform_direction)
    {

        if (signal_size_ != sequence.size())
        {
            throw std::invalid_argument(
                    "CudaFastFourierTransform can transform only data of the same size as was specified on construction.");
        }

        hipfftComplex *host_signal = CudaUtils::VectorToCufftComplex(sequence);

        hipfftResult result;

        hipLibXtDesc *device_signal;
        result = hipfftXtMalloc(cufft_execution_plan_, &device_signal, HIPFFT_XT_FORMAT_INPLACE);
        CudaUtils::checkCufftErrors(result, "CUFFT FORWARD allocation across GPUs");

        result = hipfftXtMemcpy(cufft_execution_plan_, device_signal, host_signal, HIPFFT_COPY_HOST_TO_DEVICE);
        CudaUtils::checkCufftErrors(result, "CUFFT FORWARD memory copying from Host to Device");

        std::cout << std::endl << "CUFFT FORWARD memory allocated across GPUs: " << signal_memory_size_ << " bytes." << std::endl;

        // NOTE: Transformed signal will be written instead of source signal to escape memory wasting
        clock_t execution_time = clock();
        result = hipfftXtExecDescriptorC2C(
                cufft_execution_plan_,
                device_signal,
                device_signal,
                transform_direction
        );
        std::cout << std::endl << "CUFFT FORWARD Transformation finished in: " << (float) (clock() - execution_time) / CLOCKS_PER_SEC << " seconds " << std::endl;
        CudaUtils::checkCufftErrors(result, "CUFFT FORWARD C2C execution");

        // Copy Device memory (FFT calculation results - device_signal) to Host memory (RAM)
        result = hipfftXtMemcpy(cufft_execution_plan_, host_signal, device_signal, HIPFFT_COPY_DEVICE_TO_HOST);
        CudaUtils::checkCufftErrors(result, "CUFFT FORWARD C2C Copying execution results from Device to Host");

        vector<complex<double> > result_vector = CudaUtils::CufftComplexToVector(host_signal, signal_size_);

        hipfftXtFree(device_signal);
        free(host_signal);

        return result_vector;
    }

    hipLibXtDesc* DirectTransformLibXtDesc(const vector<complex<double> > &sequence)
    {
        if (signal_size_ != sequence.size())
        {
            throw std::invalid_argument(
                    "CudaFastFourierTransform LibXtDesc can transform only data of doubled size of a signal size.");
        }

        hipfftResult result;

        hipfftComplex *host_signal = CudaUtils::VectorToCufftComplex(sequence);

        hipLibXtDesc *device_transform;
        result = hipfftXtMalloc(cufft_execution_plan_, &device_transform, HIPFFT_XT_FORMAT_INPLACE);
        CudaUtils::checkCufftErrors(result, "CUFFT FORWARD LibXtDesc allocation across GPUs");

        result = hipfftXtMemcpy(cufft_execution_plan_, device_transform, host_signal, HIPFFT_COPY_HOST_TO_DEVICE);
        CudaUtils::checkCufftErrors(result, "CUFFT FORWARD LibXtDesc memory copying from Host to Device");

        // NOTE: Transformed signal will be written instead of source signal to escape memory wasting
        clock_t execution_time = clock();
        result = hipfftXtExecDescriptorC2C(
                cufft_execution_plan_,
                device_transform,
                device_transform,
                HIPFFT_FORWARD
        );
        std::cout << std::endl << "CUFFT FORWARD LibXtDesc Transformation finished in: " << (float) (clock() - execution_time) / CLOCKS_PER_SEC << " seconds " << std::endl;
        CudaUtils::checkCufftErrors(result, "CUFFT FORWARD LibXtDesc C2C execution");

        // Copy the data to natural order on GPUs
        hipLibXtDesc *natural_ordered_transform;
        hipfftXtMalloc(cufft_execution_plan_, &natural_ordered_transform, HIPFFT_XT_FORMAT_INPLACE);
        CudaUtils::checkCufftErrors(result, "CUFFT FORWARD LibXtDesc C2C allocation memory for result");

        hipfftXtMemcpy(cufft_execution_plan_, natural_ordered_transform, device_transform, HIPFFT_COPY_DEVICE_TO_DEVICE);
        CudaUtils::checkCufftErrors(result, "CUFFT FORWARD LibXtDesc C2C memory copying from Device to Host");

        hipfftComplex *f = (hipfftComplex *) malloc (natural_ordered_transform->descriptor->size[0]);
        hipfftXtMemcpy(cufft_execution_plan_, (void **) &f, natural_ordered_transform->descriptor->data[0], HIPFFT_COPY_DEVICE_TO_HOST);
        int size = (int) natural_ordered_transform->descriptor->size[0] / sizeof(hipfftComplex);
        std::cout << std::endl << "SIZE 1: " << size << std::endl << std::endl;
        for (int i = 0; i < size; i++) {
            std::cout << f[i].x << "\t" << f[i].y << std::endl;
        }

        hipfftComplex *s = (hipfftComplex *) malloc (natural_ordered_transform->descriptor->size[1]);
        hipfftXtMemcpy(cufft_execution_plan_, (void **) &s, natural_ordered_transform->descriptor->data[1], HIPFFT_COPY_DEVICE_TO_HOST);
        size = (int) natural_ordered_transform->descriptor->size[1] / sizeof(hipfftComplex);
        std::cout << std::endl << "SIZE 2: " << size << std::endl << std::endl;
        for (int i = 0; i < size; i++) {
            std::cout << s[i].x << "\t" << s[i].y << std::endl;
        }

        hipfftXtFree(device_transform);
        return natural_ordered_transform;
    }

    // For this method it is assumed, that input_signal is already in GPU memory
    vector<complex<double> > InverseTransformLibXtDesc(hipLibXtDesc *device_signal, int signal_size)
    {
        hipfftResult result;

        clock_t execution_time = clock();

        // NOTE: Transformed signal will be written instead of source signal to escape memory wasting
        result = hipfftXtExecDescriptorC2C(
                cufft_execution_plan_,
                device_signal,
                device_signal,
                HIPFFT_BACKWARD
        );
        std::cout << std::endl << "CUFFT INVERSE Transformation finished in: " << (float) (clock() - execution_time) / CLOCKS_PER_SEC << " seconds " << std::endl;
        CudaUtils::checkCufftErrors(result, "CUFFT INVERSE LibXtDesc C2C execution");

        hipfftComplex *host_transformed = (hipfftComplex *) malloc (signal_memory_size_);
        result = hipfftXtMemcpy(cufft_execution_plan_, host_transformed, device_signal, HIPFFT_COPY_DEVICE_TO_HOST);
        CudaUtils::checkCufftErrors(result, "CUFFT INVERSE LibXtDesc C2C Copying results from Device to Host");

        vector<complex<double> > result_vector = CudaUtils::CufftComplexToVector(host_transformed, signal_size_);

        hipfftXtFree(device_signal);
        free(host_transformed);

        return result_vector;
    }

    int GetAvailableGPUs()
    {
        return available_gpus;
    }

private:

    // static fields and initializers
    static boost::mutex cuda_mutex_;

    // instance fields and initializers
    size_t signal_size_;
    int transforms_count_;
    int signal_memory_size_;

    int available_gpus;
    size_t *cufft_work_size_;
    hipfftHandle cufft_execution_plan_;

    int* GetAvailableGPUArray()
    {
        int *gpu_array = (int*) malloc(sizeof(int) * available_gpus);
        for (unsigned int index = 0; index < available_gpus; index++)
        {
            gpu_array[index] = index;
        }
        return gpu_array;
    }
};

boost::mutex CudaFastFourierTransform::CudaFastFourierTransformImpl::cuda_mutex_;

CudaFastFourierTransform::CudaFastFourierTransform(size_t sequence_size, int planned_executions)
{
    cuda_fast_fourier_transform_impl_ =
            boost::make_shared<CudaFastFourierTransformImpl>(sequence_size, planned_executions);
}

CudaFastFourierTransform::~CudaFastFourierTransform()
{
}

vector<complex<double> > CudaFastFourierTransform::DirectTransform(
        const vector<complex<double> > &sequence) const
{
    return cuda_fast_fourier_transform_impl_->DirectTransform(sequence);
}

vector<complex<double> > CudaFastFourierTransform::InverseTransform(
        const vector<complex<double> > &sequence) const
{
    return cuda_fast_fourier_transform_impl_->InverseTransform(sequence);
}

hipLibXtDesc* CudaFastFourierTransform::DirectTransformLibXtDesc(
        const vector<complex<double> > &sequence) const
{
    return cuda_fast_fourier_transform_impl_->DirectTransformLibXtDesc(sequence);
}

vector<complex<double> > CudaFastFourierTransform::InverseTransformLibXtDesc(
        hipLibXtDesc *device_signal, int signal_size) const
{
    return cuda_fast_fourier_transform_impl_->InverseTransformLibXtDesc(device_signal, signal_size);
}

int CudaFastFourierTransform::GetAvailableGPUs() const {
    return cuda_fast_fourier_transform_impl_->GetAvailableGPUs();
}

#endif //BUILD_WITH_CUDA

} // namespace dsp
} // namespace k52