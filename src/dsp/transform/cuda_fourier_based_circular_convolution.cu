#include "hip/hip_runtime.h"
#include <k52/dsp/transform/cuda_fourier_based_circular_convolution.h>
#include <cstdio>
#include <stdexcept>

#ifdef BUILD_WITH_CUDA

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <boost/thread/mutex.hpp>
#include <boost/smart_ptr/make_shared.hpp>
#include <k52/dsp/transform/util/cuda_utils.h>
#include "../../../../../../../usr/local/cuda/include/hip/hip_runtime_api.h"
#include "../../../../../../../usr/local/cuda/include/"
#include "../../../../../../../usr/local/cuda/include/hipfft/hipfftXt.h"

// TODO: DELETE THIS IMPORTS - THEY ARE ONLY FOR CLION COMPILATION PURPOSE

#endif

using ::std::vector;
using ::std::complex;
using ::k52::dsp::CudaFourierBasedCircularConvolution;

#ifdef BUILD_WITH_CUDA

// CUDA kernel function used to multiply two signals in parallel
// NOTE: Result is written instead of first signal
__global__ void MultiplySignals(hipfftComplex *first,
                                hipfftComplex *second,
                                int signal_size)
{
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < signal_size)
    {
        // Elements of the result of signals multiplication are calculated in parallel
        // using thread_id variable - thread index.
        // Each thread calculates one element of result sequence at first[thread_id] moment.
        hipfftComplex result_element;
        result_element.x = first[thread_id].x * second[thread_id].x - first[thread_id].y * second[thread_id].y;
        result_element.y = first[thread_id].x * second[thread_id].y + first[thread_id].y * second[thread_id].x;
        first[thread_id] = result_element;
    }
}

CudaFourierBasedCircularConvolution::CudaFourierBasedCircularConvolution(size_t sequence_size, int batch_size)
{
    cufft_transformer_ = boost::make_shared<CudaFastFourierTransform>(sequence_size, batch_size);
}

vector<complex<double> > CudaFourierBasedCircularConvolution::EvaluateConvolution(
        const vector<complex<double> > &first_signal,
        const vector<complex<double> > &second_signal) const
{
    if (first_signal.size() != second_signal.size())
    {
        throw std::runtime_error("Can evaluate convolution only for sequences of the same size.");
    }

    size_t signal_size = first_signal.size();

    // Create one signal based on input signals to pass it through
    // Two GPUs via cudaLibXt, assuming, that signals are of the same size
    // And sum of sizes is multiplied by 2.
    vector<complex<double> > sum_signal;

    sum_signal.reserve(signal_size * 2);
    copy(first_signal.begin(), first_signal.end(), back_inserter(sum_signal));
    copy(second_signal.begin(), second_signal.end(), back_inserter(sum_signal));

    // Here are used additional CudaFastFourierTransform methods
    // to prevent from useless copying hipfftComplex arrays into vector
    hipLibXtDesc *sum_signal_transform =
            cufft_transformer_->DirectTransformLibXtDesc(sum_signal);

    int signal_memory_size = sizeof(hipfftComplex) * signal_size;

    cudaXtDesc *result_descriptor = sum_signal_transform->descriptor;

    // Get FFT-results from each GPU
    hipfftComplex *gpu0_result = (hipfftComplex*) (result_descriptor->data[0]);
    hipfftComplex *gpu1_result = (hipfftComplex*) (result_descriptor->data[1]);

    // Copy FFT-results from GPU_1 to GPU_0
    // To calculate multiplication in parallel on one device
    hipfftComplex *gpu0_result_from_gpu1;
    hipMalloc((void**) &gpu0_result_from_gpu1, signal_memory_size);
    hipMemcpy(gpu0_result_from_gpu1, gpu1_result, signal_memory_size, hipMemcpyDeviceToDevice);

    hipSetDevice(0);
    MultiplySignals<<<64, 256>>>(gpu0_result_from_gpu1, gpu0_result, signal_size);

    for (int i = 0; i < signal_size; i++) {
        std::cout << gpu0_result_from_gpu1[i].x << "\t" << gpu0_result_from_gpu1[i].y << std::endl;
    }

    hipfftXtFree(sum_signal_transform);
    hipFree(gpu0_result);

    return cufft_transformer_->InverseTransformFromDevice(gpu0_result_from_gpu1, signal_size);
}

#endif //BUILD_WITH_CUDA
