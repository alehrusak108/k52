#include <k52/dsp/transform/cuda_fast_fourier_transform.h>
#include <k52/dsp/transform/util/cuda_utils.h>
#include <cstdio>
#include <stdexcept>

#ifdef BUILD_WITH_CUDA

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <boost/thread/mutex.hpp>
#include <boost/smart_ptr/make_shared.hpp>
#include "../../../../../../../usr/local/cuda/include/hipfft/hipfft.h"
#include "../../../../../../../usr/local/cuda/include/hipfft/hipfftXt.h"
#include "../../../../../../../usr/local/cuda/include/hip/hip_runtime_api.h"

// TODO: DELETE THIS IMPORTS!

#endif

using ::std::vector;
using ::std::complex;
using ::std::invalid_argument;
using ::std::runtime_error;

namespace k52
{
namespace dsp
{

#ifdef BUILD_WITH_CUDA

// Using pImpl approach to hide CUFFT from outside use
// NOTE: Prefix "device_" means that variable is allocated in CUDA Device Memory
//       Prefix "host_" means that variable is allocated in RAM (Host)

class CudaFastFourierTransform::CudaFastFourierTransformImpl
{

public:
    CudaFastFourierTransformImpl(size_t sequence_size, int executions_planned)
            : signal_size_(sequence_size), executions_planned_(executions_planned) {

        boost::mutex::scoped_lock scoped_lock(cuda_mutex_);

        if (sequence_size <= 0) {
            throw std::invalid_argument("sequence_size <= 0");
        }

        signal_memory_size_ = sizeof(hipfftComplex) * signal_size_;

        /*int nGPUs = 2, whichGPUs[2] = {0, 1};
        hipfftCreate(&cufft_execution_plan_);
        hipfftXtSetGPUs(cufft_execution_plan_, nGPUs, whichGPUs);
        hipfftMakePlan1d(&cufft_execution_plan_, signal_size_, hipfftType type, executions_planned_, size_t *workSize);
*/

        size_t work_size[1];
        hipfftCreate(&cufft_execution_plan_);

        int dimensions = 1; // 1D FFTs
        int ranks_array[] = { signal_size_ }; // Sizes of arrays of each dimension
        int istride = 1; // Distance between two successive input elements
        int ostride = 1; // Same for the output elements
        int idist = 1; // Distance between batches
        int odist = 1; // Same for the output elements
        int *inembed = NULL; // Input size with pitch (ignored for 1D transforms)
        int *onembed = NULL; // Output size with pitch (ignored for 1D transforms)

        // Single-Dimensional FFT execution plan configuration
        hipfftResult plan_prepare_result = hipfftMakePlanMany(
                cufft_execution_plan_,
                dimensions,
                ranks_array,
                inembed, istride, idist,
                onembed, ostride, odist,
                HIPFFT_C2C,
                executions_planned_,
                work_size
        );

        //hipfftResult plan_prepare_result = hipfftPlan1d(&cufft_execution_plan_, signal_size_, HIPFFT_C2C, 1);
        std::cout << std::endl << "CUFFT Execution Plan prepared: " << plan_prepare_result << std::endl;
    }

    ~CudaFastFourierTransformImpl() {

        std::cout << "Destroying CUFFT Context..." << std::endl;

        // Destroy CUFFT Execution Plan
        hipfftResult destructor_result = hipfftDestroy(cufft_execution_plan_);
        std::cout << "CUFFT Execution Plan destructor returned: " << destructor_result << std::endl;

        boost::mutex::scoped_lock scoped_lock(cuda_mutex_);
    }

    vector<complex<double> > DirectTransform(const vector<complex<double> > &sequence)
    {
        return Transform(sequence, HIPFFT_FORWARD);
    }

    vector<complex<double> > InverseTransform(const vector<complex<double> > &sequence)
    {
        return Transform(sequence, HIPFFT_BACKWARD);
    }

    vector<complex<double> > Transform(const vector<complex<double> > &sequence, int transform_direction) const
    {

        if (signal_size_ != sequence.size()) {
            throw std::invalid_argument(
                    "CudaFastFourierTransform can transform only data of the same size as was specified on construction.");
        }

        hipfftComplex *host_signal = CudaUtils::VectorToCufftComplex(sequence);

        // Copy sequence data memory to device
        hipfftComplex *device_signal;
        hipMalloc((void**) &device_signal, signal_memory_size_);
        hipMemcpy(device_signal, host_signal, signal_memory_size_, hipMemcpyHostToDevice);
        std::cout << std::endl << "Signal memory allocated: " << signal_memory_size_ << " bytes." << std::endl;

        // NOTE: Transformed signal will be written instead of source signal to escape memory wasting
        hipfftResult execution_result = hipfftExecC2C(cufft_execution_plan_, device_signal, device_signal, transform_direction);
        std::cout << std::endl << "CUFFT C2C (float) Execution result: " << execution_result << std::endl;

        // Copy Device memory (FFT calculation results - d_signal_output_) to Host memory (RAM)
        hipfftComplex *host_result = (hipfftComplex *) malloc(signal_memory_size_);
        hipMemcpy(host_result, device_signal, signal_memory_size_, hipMemcpyDeviceToHost);

        vector<complex<double> > result_vector = CudaUtils::CufftComplexToVector(host_result, signal_size_);

        hipFree(device_signal);
        hipFree(host_result);

        return result_vector;
    }

private:

    // static fields and initializers
    static boost::mutex cuda_mutex_;

    // instance fields and initializers
    size_t signal_size_;
    int executions_planned_;
    int signal_memory_size_;

    hipfftHandle cufft_execution_plan_;
};

boost::mutex CudaFastFourierTransform::CudaFastFourierTransformImpl::cuda_mutex_;

CudaFastFourierTransform::CudaFastFourierTransform(size_t sequence_size, int planned_executions)
{
    cuda_fast_fourier_transform_impl_ =
            boost::make_shared<CudaFastFourierTransformImpl>(sequence_size, planned_executions);
}

CudaFastFourierTransform::~CudaFastFourierTransform()
{
}

vector<complex<double> > CudaFastFourierTransform::DirectTransform(
        const vector<complex<double> > &sequence) const
{
    return cuda_fast_fourier_transform_impl_->DirectTransform(sequence);
}

vector<complex<double> > CudaFastFourierTransform::InverseTransform(
        const vector<complex<double> > &sequence) const
{
    return cuda_fast_fourier_transform_impl_->InverseTransform(sequence);
}

        // Planned Execitions (batch) other than 1 for hipfftPlan1d() have been deprecated.
        // Here used hipfftPlanMany() for multiple execution.

        /*int dimensions = 1; // 1D FFTs
        int ranks_array[] = { signal_size_ }; // Sizes of arrays of each dimension
        int istride = executions_planned_; // Distance between two successive input elements
        int ostride = executions_planned_; // Same for the output elements
        int idist = 1; // Distance between batches
        int odist = 1; // Same for the output elements
        int inembed[] = { 0 }; // Input size with pitch (ignored for 1D transforms)
        int onembed[] = { 0 }; // Output size with pitch (ignored for 1D transforms)

        // Single-Dimensional FFT execution plan configuration
        hipfftResult plan_prepare_result = hipfftPlanMany(
                &cufft_execution_plan_,
                dimensions,
                ranks_array,
                inembed, istride, idist,
                onembed, ostride, odist,
                HIPFFT_C2C,
                executions_planned_
        );*/

#endif //BUILD_WITH_CUDA

} // namespace dsp
} // namespace k52