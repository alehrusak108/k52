#include "hip/hip_runtime.h"
#include <k52/dsp/transform/fast_fourier_transform.h>

#include <iostream>
#include <ctime>
#include <cstdlib>
#include <vector>
#include <fstream>

#ifdef BUILD_WITH_CUDA

#include <k52/dsp/transform/cuda_fast_fourier_transform.h>

#endif

#define CUFFT_EXECUTIONS_PLANNED 1

using namespace std;

using namespace k52::dsp;

double CUFFTPerformanceTest(vector<complex<double> > &input_signal) {

    ofstream test_output;
    test_output.open("test_output.txt", ios::out | ios::app);
    test_output << endl << "[ CUFFT Performance TEST ] STARTED." << endl;

    clock_t planning_time = clock();

    CudaFastFourierTransform cufftTransformer(input_signal.size(), CUFFT_EXECUTIONS_PLANNED);

    test_output << "CUFFT Execution Plan prepared in: " << (float) (clock() - planning_time) / CLOCKS_PER_SEC << " seconds" << endl;

    clock_t execution_time = clock();

    // In this test, we don't care about transformation result
    vector<complex<double> > output = cufftTransformer.DirectTransform(input_signal);

    /*cout << endl << "CUFFT OUTPUT" << endl;
    for (int i = 0; i < input_signal.size(); i++) {
        cout << output[i].real() << "\t\t" << output[i].imag() << endl;
    }*/

    clock_t finish = clock() - execution_time;
    test_output << endl << "Time elapsed for CUFFT Transform Test: " << (double) finish / CLOCKS_PER_SEC << " seconds " << endl << endl;
    test_output << "[ CUFFT Performance TEST ] FINISHED." << endl << endl;
    test_output.close();
    return (double) finish / CLOCKS_PER_SEC;
}

double FFTWPerformanceTest(vector<complex<double> > &input_signal) {

    ofstream test_output;
    test_output.open("test_output.txt", ios::out | ios::app);
    test_output << "[ FFTW3 Performance TEST ] STARTED." << endl;

    clock_t planning_time = clock();

    FastFourierTransform fftw3Transformer(input_signal.size());

    test_output << endl << "FFTW3 Execution Plan prepared in: " << (float) (clock() - planning_time) / CLOCKS_PER_SEC << " seconds" << endl;

    clock_t execution_time = clock();

    // In this test, we don't care about transformation result
    vector<complex<double> > output = fftw3Transformer.DirectTransform(input_signal);

    /*cout << endl << "FFTW OUTPUT" << endl;
    for (int i = 0; i < input_signal.size(); i++) {
        cout << output[i].real() << "\t" << output[i].imag() << endl;
    }*/

    clock_t finish = clock() - execution_time;
    test_output << endl << "Time elapsed for FFTW3 Transform Test: " << (double) finish / CLOCKS_PER_SEC << " seconds " << endl << endl;
    test_output << "[ FFTW3 Performance TEST ] FINISHED." << endl << endl;
    test_output.close();
    return (double) finish / CLOCKS_PER_SEC;
}

vector<complex<double> > PrepareTestSignal(size_t signal_size) {
    vector<complex<double> > input_signal(signal_size);
    for (size_t index = 0; index < signal_size; index++) {
        //input_signal[index].real(index);
        //input_signal[index].imag(0);
        input_signal[index].real(-5 + rand() % 15);
        input_signal[index].imag(-5 + rand() % 15);
    }
    /*for (int i = 0; i < signal_size; i++) {
        cout << input_signal[i].real() << "\t" << input_signal[i].imag() << endl;
    }*/
    return input_signal;
}

int main(int argc, char* argv[])
{
    srand(time(NULL));
    ofstream test_output;
    test_output.open("test_output.txt", ios::out | ios::app);
    int signal_size = 33554432;
    int window_size = 1024;
    int windows_count = 33554432 / 1024;
    vector<complex<double> > input_signal = PrepareTestSignal(signal_size);
    double cufft_summary = 0.0;
    double fftw_summary = 0.0;
    for (int index = 0; index < windows_count - 1; index++) {
        vector<complex<double> >::const_iterator start = input_signal.begin() + index * window_size;
        vector<complex<double> >::const_iterator end = input_signal.begin() + (index + 1) * window_size;
        vector<complex<double> > window(start, end);
        //test_output << endl << "TEST #" << test_number << "\t" << "Signal Length is: " << signal_size << endl;
        cufft_summary += CUFFTPerformanceTest(window);
        //test_output << "---------------------------------------------" << endl << endl;
        fftw_summary += FFTWPerformanceTest(window);
        //test_output << "===============================================================================" << endl << endl;
        //signal_size *= 2;
    }
    test_output << endl << endl << "CUFFT SUMMARY TIME: " << cufft_summary << endl << endl;
    test_output << endl << endl << "FFTW3 SUMMARY TIME: " << fftw_summary << endl << endl;
    test_output.close();
}
