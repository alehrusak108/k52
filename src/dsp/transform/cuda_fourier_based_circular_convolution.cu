#include "hip/hip_runtime.h"
#include <k52/dsp/transform/cuda_fourier_based_circular_convolution.h>
#include <cstdio>
#include <stdexcept>

#ifdef BUILD_WITH_CUDA

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <boost/thread/mutex.hpp>
#include <boost/smart_ptr/make_shared.hpp>
#include <k52/dsp/transform/util/cuda_utils.h>
#include "../../../../../../../usr/local/cuda/include/hip/hip_runtime_api.h"
#include "../../../../../../../usr/local/cuda/include/"

// TODO: DELETE THIS IMPORTS - THEY ARE ONLY FOR CLION COMPILATION PURPOSE

#endif

using ::std::vector;
using ::std::complex;
using ::k52::dsp::CudaFourierBasedCircularConvolution;

#ifdef BUILD_WITH_CUDA

// CUDA kernel function used to multiply two signals in parallel
// Result is written instead of first signal
__global__ void MultiplySignals(hipfftComplex *first,
                                hipfftComplex *second,
                                int signal_size)
{
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < signal_size)
    {
        // Elements of the result of signals multiplication are calculated in parallel
        // using thread_id variable - thread index.
        // Each thread calculates one element of result sequence at first[thread_id] moment.
        hipfftComplex result_element;
        result_element.x = first[thread_id].x * second[thread_id].x - first[thread_id].y * second[thread_id].y;
        result_element.y = first[thread_id].x * second[thread_id].y + first[thread_id].y * second[thread_id].x;
        first[thread_id] = result_element;
    }
}

CudaFourierBasedCircularConvolution::CudaFourierBasedCircularConvolution(size_t sequence_size, int batch_size)
{
    cufft_transformer_ = boost::make_shared<CudaFastFourierTransform>(sequence_size, batch_size);
}

vector<complex<double> > CudaFourierBasedCircularConvolution::EvaluateConvolution(
        const vector<complex<double> > &first_sequence,
        const vector<complex<double> > &second_sequence) const
{
    if (first_sequence.size() != second_sequence.size())
    {
        throw std::runtime_error("Can evaluate convolution only for sequences of the same size.");
    }

    size_t signal_size = first_sequence.size();

    // Here are used additional CudaFastFourierTransform methods
    // to prevent from useless copying hipfftComplex arrays into vector
    hipLibXtDesc *first_sequence_transform =
            cufft_transformer_->DirectTransformMemoryDesc(first_sequence);
    hipLibXtDesc *second_sequence_transform =
            cufft_transformer_->DirectTransformMemoryDesc(second_sequence);

    int signal_memory_size = sizeof(hipfftComplex) * signal_size;

    // Copy transformed signal from first device to zero one
    // To multiply them on one device
    hipfftComplex *gpu1_transform = (hipfftComplex*) (first_sequence_transform->descriptor->data[1]);
    //hipfftComplex *gpu0_transform = (hipfftComplex*) (->descriptor->data[0]);
    hipSetDevice(0);
    hipfftComplex *gpu0_transform_from_gpu1;
    hipMalloc((void**) &gpu0_transform_from_gpu1, signal_memory_size);

    hipMemcpy(gpu0_transform_from_gpu1, gpu1_transform, signal_memory_size, hipMemcpyDeviceToDevice);

    //MultiplySignals<<<64, 256>>>(gpu0_transform_from_gpu1, d_second, d_multiplication, signal_size);

    //vector<complex<double> > convolution = CudaUtils::CufftComplexToVector(d_multiplication, signal_size);

    return first_sequence;
}

#endif //BUILD_WITH_CUDA
